#include "hip/hip_runtime.h"
#include "core.h"

extern void cudaCallbackCPU(int k, int m, int n, float* searchPoints,
                            float* referencePoints, int** results)
{

    int* tmp = (int*)malloc(sizeof(int) * m);
    int minIndex;
    float minSquareSum, diff, squareSum;

    // Iterate over all search points
    for (int mInd = 0; mInd < m; mInd++) {
        minSquareSum = -1;
        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            squareSum = 0;
            for (int kInd = 0; kInd < k; kInd++) {
                diff = searchPoints[k * mInd + kInd] - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        tmp[mInd] = minIndex;
    }

    *results = tmp;
    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}



// searchParallel
__global__ void kernel1(int k, int m, int n, float* searchPoints,
                        float* referencePoints, int* indices)
{
    int minIndex;
    float minSquareSum, diff, squareSum;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < m) {
        minSquareSum = -1;
        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            squareSum = 0;
            for (int kInd = 0; kInd < k; kInd++) {
                diff = searchPoints[k * tid + kInd] - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        indices[tid] = minIndex;
    }
}

extern void cudaCallback1(int k, int m, int n, float* searchPoints,
                          float* referencePoints, int** results)
{

    int block_size = divup(m, 1024);

    int* indices_h = (int*)malloc(sizeof(int) * m);

    int* indices_d;
    float* searchPoints_d;
    float* referencePoints_d;

    // Memory allocation
    CHECK(hipMalloc((void**)(&indices_d), sizeof(int) * m));
    CHECK(hipMalloc((void**)(&searchPoints_d), sizeof(float) * k * m));
    CHECK(hipMalloc((void**)(&referencePoints_d), sizeof(float) * k * n));

    // Memory Copy : Host to Device
    CHECK(hipMemcpy((void*)(indices_d), (void*)(indices_h), sizeof(int) * m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)(searchPoints_d), (void*)(searchPoints), sizeof(float) * k * m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)(referencePoints_d), (void*)(referencePoints), sizeof(float) * k * n, hipMemcpyHostToDevice));


    kernel1 <<< block_size, 1024>>> (k, m, n, searchPoints_d, referencePoints_d, indices_d);

    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy((void*)(indices_h), (void*)(indices_d), sizeof(int) * m, hipMemcpyDeviceToHost));


    *results = indices_h;

    hipFree((void*)indices_d);
    hipFree((void*)searchPoints_d);
    hipFree((void*)referencePoints_d);

    indices_h = NULL;
    indices_d = NULL;
    searchPoints_d = NULL;
    referencePoints_d = NULL;

    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}


// referenceParallel
__global__ void kernel2(int k, int n, float* sub_searchPoints,
                        float* referencePoints, float* dist)
{
    float diff, squareSum;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        squareSum = 0;
        for (int i = 0; i < k; i++) {
            diff = sub_searchPoints[i] - referencePoints[k * tid + i];
            squareSum += (diff * diff);
        }
        dist[tid] = squareSum;
    }
}

extern void cudaCallback2(int k, int m, int n, float* searchPoints,
                          float* referencePoints, int** results)
{
    int minIndex;
    float minSquareSum;

    int block_size = divup(n, 1024);

    int* tmp = (int*)malloc(sizeof(int) * m);
    float* dist_h = (float*)malloc(sizeof(float) * n);
    float* sub_searchPoints_h = (float*)malloc(sizeof(float) * k);

    float* referencePoints_d;
    float* dist_d;
    float* sub_searchPoints_d;

    // Memory allocation
    CHECK(hipMalloc((void**)(&referencePoints_d), sizeof(float) * k * n));
    CHECK(hipMalloc((void**)(&dist_d), sizeof(float) * n));
    CHECK(hipMalloc((void**)(&sub_searchPoints_d), sizeof(float) * k));

    // Memory Copy : Host to Device
    CHECK(hipMemcpy((void*)(referencePoints_d), (void*)(referencePoints), sizeof(float) * k * n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)(dist_d), (void*)(dist_h), sizeof(float) * n, hipMemcpyHostToDevice));


    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            sub_searchPoints_h[j] = searchPoints[i * k + j];
            // printf("%f ", sub_searchPoints_h[j]);
        }
        // getchar();

        // Memory Copy : Host to Device
        CHECK(hipMemcpy((void*)(sub_searchPoints_d), (void*)(sub_searchPoints_h), sizeof(float) * k, hipMemcpyHostToDevice));

        kernel2 <<< block_size, 1024>>> (k, n, sub_searchPoints_d, referencePoints_d, dist_d);

        CHECK(hipDeviceSynchronize());

        // Memory Copy : Device to Host
        CHECK(hipMemcpy((void*)(dist_h), (void*)(dist_d), sizeof(float) * n, hipMemcpyDeviceToHost));

        minSquareSum = -1;
        for (int j = 0; j < n; j++) {
            if (minSquareSum < 0 || minSquareSum > dist_h[j]) {
                minSquareSum = dist_h[j];
                minIndex = j;
            }
        }

        tmp[i] = minIndex;
    }

    *results = tmp;

    free(dist_h);
    free(sub_searchPoints_h);

    hipFree((void*)referencePoints_d);
    hipFree((void*)dist_d);
    hipFree((void*)sub_searchPoints_d);

    dist_h = NULL;
    sub_searchPoints_h = NULL;
    referencePoints_d = NULL;
    dist_d = NULL;
    sub_searchPoints_d = NULL;

    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}


// combinedParallel
__global__ void RPkernel(int k, int n, float* sub_searchPoints,
                         float* referencePoints, float* dist)
{
    float diff, squareSum;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        squareSum = 0;
        for (int i = 0; i < k; i++) {
            diff = sub_searchPoints[i] - referencePoints[k * tid + i];
            squareSum += (diff * diff);
        }
        dist[tid] = squareSum;
    }
}

__global__ void SPkernel(int k, int m, int n, float* searchPoints,
                         float* referencePoints, int* indices)
{
    int minIndex;
    float minSquareSum, diff, squareSum;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < m) {
        minSquareSum = -1;
        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            squareSum = 0;
            for (int kInd = 0; kInd < k; kInd++) {
                diff = searchPoints[k * tid + kInd] - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        indices[tid] = minIndex;
    }
}

extern void cudaCallback3(int k, int m, int n, float* searchPoints,
                          float* referencePoints, int** results)
{
    if (m >= 1024) {
        int block_size = divup(m, 1024);

        int* indices_h = (int*)malloc(sizeof(int) * m);

        int* indices_d;
        float* searchPoints_d;
        float* referencePoints_d;

        // Memory allocation
        CHECK(hipMalloc((void**)(&indices_d), sizeof(int) * m));
        CHECK(hipMalloc((void**)(&searchPoints_d), sizeof(float) * k * m));
        CHECK(hipMalloc((void**)(&referencePoints_d), sizeof(float) * k * n));

        // Memory Copy : Host to Device
        CHECK(hipMemcpy((void*)(indices_d), (void*)(indices_h), sizeof(int) * m, hipMemcpyHostToDevice));
        CHECK(hipMemcpy((void*)(searchPoints_d), (void*)(searchPoints), sizeof(float) * k * m, hipMemcpyHostToDevice));
        CHECK(hipMemcpy((void*)(referencePoints_d), (void*)(referencePoints), sizeof(float) * k * n, hipMemcpyHostToDevice));


        SPkernel <<< block_size, 1024>>> (k, m, n, searchPoints_d, referencePoints_d, indices_d);

        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy((void*)(indices_h), (void*)(indices_d), sizeof(int) * m, hipMemcpyDeviceToHost));


        *results = indices_h;

        hipFree((void*)indices_d);
        hipFree((void*)searchPoints_d);
        hipFree((void*)referencePoints_d);

        indices_h = NULL;
        indices_d = NULL;
        searchPoints_d = NULL;
        referencePoints_d = NULL;
    }
    else {
        int minIndex;
        float minSquareSum;

        int block_size = divup(n, 1024);

        int* tmp = (int*)malloc(sizeof(int) * m);
        float* dist_h = (float*)malloc(sizeof(float) * n);
        float* sub_searchPoints_h = (float*)malloc(sizeof(float) * k);

        float* referencePoints_d;
        float* dist_d;
        float* sub_searchPoints_d;

        // Memory allocation
        CHECK(hipMalloc((void**)(&referencePoints_d), sizeof(float) * k * n));
        CHECK(hipMalloc((void**)(&dist_d), sizeof(float) * n));
        CHECK(hipMalloc((void**)(&sub_searchPoints_d), sizeof(float) * k));

        // Memory Copy : Host to Device
        CHECK(hipMemcpy((void*)(referencePoints_d), (void*)(referencePoints), sizeof(float) * k * n, hipMemcpyHostToDevice));
        CHECK(hipMemcpy((void*)(dist_d), (void*)(dist_h), sizeof(float) * n, hipMemcpyHostToDevice));


        for (int i = 0; i < m; i++) {
            for (int j = 0; j < k; j++) {
                sub_searchPoints_h[j] = searchPoints[i * k + j];
                // printf("%f ", sub_searchPoints_h[j]);
            }
            // getchar();

            // Memory Copy : Host to Device
            CHECK(hipMemcpy((void*)(sub_searchPoints_d), (void*)(sub_searchPoints_h), sizeof(float) * k, hipMemcpyHostToDevice));

            RPkernel <<< block_size, 1024>>> (k, n, sub_searchPoints_d, referencePoints_d, dist_d);

            CHECK(hipDeviceSynchronize());

            // Memory Copy : Device to Host
            CHECK(hipMemcpy((void*)(dist_h), (void*)(dist_d), sizeof(float) * n, hipMemcpyDeviceToHost));

            minSquareSum = -1;
            for (int j = 0; j < n; j++) {
                if (minSquareSum < 0 || minSquareSum > dist_h[j]) {
                    minSquareSum = dist_h[j];
                    minIndex = j;
                }
            }

            tmp[i] = minIndex;
        }

        *results = tmp;

        free(dist_h);
        free(sub_searchPoints_h);

        hipFree((void*)referencePoints_d);
        hipFree((void*)dist_d);
        hipFree((void*)sub_searchPoints_d);

        dist_h = NULL;
        sub_searchPoints_h = NULL;
        referencePoints_d = NULL;
        dist_d = NULL;
        sub_searchPoints_d = NULL;
    }
    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}


// dynamicParallel
__global__ void kernel4(int k, int n, int bias, float* searchPoints,
                        float* referencePoints, float* dist)
{
    float diff, squareSum;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        squareSum = 0;
        for (int i = 0; i < k; i++) {
            diff = searchPoints[k * bias + i] - referencePoints[k * tid + i];
            squareSum += (diff * diff);
        }
        dist[bias * n + tid] = squareSum;
    }
}


__global__ void over_kernel4(int k, int m, int n, int block_size,
                             float* searchPoints, float* referencePoints,
                             float* dist_d, int* res)
{

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < m) {
        int minIndex;
        float minSquareSum;

        kernel4 <<< block_size, 1024>>> (k, n, tid, searchPoints, referencePoints, dist_d);
        
        hipDeviceSynchronize();
        
        minSquareSum = -1;
        for (int j = 0; j < n; j++) {
            if (minSquareSum < 0 || minSquareSum > dist_d[tid * n + j]) {
                minSquareSum = dist_d[tid * n + j];
                minIndex = j;
            }
        }
        res[tid] = minIndex;
    }
}


extern void cudaCallback4(int k, int m, int n, float* searchPoints,
                          float* referencePoints, int** results)
{
    int block_size1 = divup(m, 1024);
    int block_size2 = divup(n, 1024);
    
    int* tmp_h = (int*)malloc(sizeof(int) * m);

    int* tmp_d;
    float* searchPoints_d;
    float* referencePoints_d;
    float* dist_d;

    // Memory allocation
    CHECK(hipMalloc((void**)(&tmp_d), sizeof(int) * m));
    CHECK(hipMalloc((void**)(&dist_d), sizeof(float) * m * n));
    CHECK(hipMalloc((void**)(&searchPoints_d), sizeof(float) * k * m));
    CHECK(hipMalloc((void**)(&referencePoints_d), sizeof(float) * k * n));


    // Memory Copy : Host to Device
    CHECK(hipMemcpy((void*)(tmp_d), (void*)(tmp_h), sizeof(int) * m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)(searchPoints_d), (void*)(searchPoints), sizeof(float) * k * m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy((void*)(referencePoints_d), (void*)(referencePoints), sizeof(float) * k * n, hipMemcpyHostToDevice));

    over_kernel4 <<< block_size1, 1024>>>(k, m, n, block_size2, searchPoints_d, referencePoints_d, dist_d, tmp_d);

    
    CHECK(hipDeviceSynchronize());

    // Memory Copy : Device to Host
    CHECK(hipMemcpy((void*)(tmp_h), (void*)(tmp_d), sizeof(int) * m, hipMemcpyDeviceToHost));
    
    *results = tmp_h;
    
    hipFree((void*)tmp_d);
    hipFree((void*)dist_d);
    hipFree((void*)searchPoints_d);
    hipFree((void*)referencePoints_d);

    tmp_d = NULL;
    dist_d = NULL;
    referencePoints_d = NULL;
    searchPoints_d = NULL;

    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}