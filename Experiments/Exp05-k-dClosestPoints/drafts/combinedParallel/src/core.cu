#include "hip/hip_runtime.h"
#include "core.h"


__global__ void RPkernel(int k, int n, float* sub_searchPoints,
                         float* referencePoints, float* dist)
{
    float diff, squareSum;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        squareSum = 0;
        for (int i = 0; i < k; i++) {
            diff = sub_searchPoints[i] - referencePoints[k * tid + i];
            squareSum += (diff * diff);
        }
        dist[tid] = squareSum;
    }
}


__global__ void SPkernel(int k, int m, int n, float* searchPoints,
                         float* referencePoints, int* indices)
{
    int minIndex;
    float minSquareSum, diff, squareSum;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < m) {
        minSquareSum = -1;
        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            squareSum = 0;
            for (int kInd = 0; kInd < k; kInd++) {
                diff = searchPoints[k * tid + kInd] - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }
            if (minSquareSum < 0 || squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        indices[tid] = minIndex;
    }
}

extern void cudaCallback(int k, int m, int n, float* searchPoints,
                         float* referencePoints, int** results)
{
    if (m >= 1024) {
        int block_size = divup(m, 1024);

        int* indices_h = (int*)malloc(sizeof(int) * m);

        int* indices_d;
        float* searchPoints_d;
        float* referencePoints_d;

        // Memory allocation
        CHECK(hipMalloc((void**)(&indices_d), sizeof(int) * m));
        CHECK(hipMalloc((void**)(&searchPoints_d), sizeof(float) * k * m));
        CHECK(hipMalloc((void**)(&referencePoints_d), sizeof(float) * k * n));

        // Memory Copy : Host to Device
        CHECK(hipMemcpy((void*)(indices_d), (void*)(indices_h), sizeof(int) * m, hipMemcpyHostToDevice));
        CHECK(hipMemcpy((void*)(searchPoints_d), (void*)(searchPoints), sizeof(float) * k * m, hipMemcpyHostToDevice));
        CHECK(hipMemcpy((void*)(referencePoints_d), (void*)(referencePoints), sizeof(float) * k * n, hipMemcpyHostToDevice));


        SPkernel <<< block_size, 1024>>> (k, m, n, searchPoints_d, referencePoints_d, indices_d);

        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy((void*)(indices_h), (void*)(indices_d), sizeof(int) * m, hipMemcpyDeviceToHost));


        *results = indices_h;

        hipFree((void*)indices_d);
        hipFree((void*)searchPoints_d);
        hipFree((void*)referencePoints_d);

        indices_h = NULL;
        indices_d = NULL;
        searchPoints_d = NULL;
        referencePoints_d = NULL;
    }
    else {
        int minIndex;
        float minSquareSum;

        int block_size = divup(n, 1024);

        int* tmp = (int*)malloc(sizeof(int) * m);
        float* dist_h = (float*)malloc(sizeof(float) * n);
        float* sub_searchPoints_h = (float*)malloc(sizeof(float) * k);

        float* referencePoints_d;
        float* dist_d;
        float* sub_searchPoints_d;

        // Memory allocation
        CHECK(hipMalloc((void**)(&referencePoints_d), sizeof(float) * k * n));
        CHECK(hipMalloc((void**)(&dist_d), sizeof(float) * n));
        CHECK(hipMalloc((void**)(&sub_searchPoints_d), sizeof(float) * k));

        // Memory Copy : Host to Device
        CHECK(hipMemcpy((void*)(referencePoints_d), (void*)(referencePoints), sizeof(float) * k * n, hipMemcpyHostToDevice));
        CHECK(hipMemcpy((void*)(dist_d), (void*)(dist_h), sizeof(float) * n, hipMemcpyHostToDevice));


        for (int i = 0; i < m; i++) {
            for (int j = 0; j < k; j++) {
                sub_searchPoints_h[j] = searchPoints[i * k + j];
                // printf("%f ", sub_searchPoints_h[j]);
            }
            // getchar();

            // Memory Copy : Host to Device
            CHECK(hipMemcpy((void*)(sub_searchPoints_d), (void*)(sub_searchPoints_h), sizeof(float) * k, hipMemcpyHostToDevice));

            RPkernel <<< block_size, 1024>>> (k, n, sub_searchPoints_d, referencePoints_d, dist_d);

            CHECK(hipDeviceSynchronize());

            // Memory Copy : Device to Host
            CHECK(hipMemcpy((void*)(dist_h), (void*)(dist_d), sizeof(float) * n, hipMemcpyDeviceToHost));

            minSquareSum = -1;
            for (int j = 0; j < n; j++) {
                if (minSquareSum < 0 || minSquareSum > dist_h[j]) {
                    minSquareSum = dist_h[j];
                    minIndex = j;
                }
            }

            tmp[i] = minIndex;
        }

        *results = tmp;

        free(dist_h);
        free(sub_searchPoints_h);

        hipFree((void*)referencePoints_d);
        hipFree((void*)dist_d);
        hipFree((void*)sub_searchPoints_d);

        dist_h = NULL;
        sub_searchPoints_h = NULL;
        referencePoints_d = NULL;
        dist_d = NULL;
        sub_searchPoints_d = NULL;
    }
    // Note that you don't have to free searchPoints, referencePoints, and
    // *results by yourself
}